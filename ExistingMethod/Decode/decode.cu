#include "hip/hip_runtime.h"
#include <stdio.h>

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

inline void assertSuccessfulAllocation(void *ptr, const char * str) {
	if (ptr == 0) {
		fprintf(stderr, "Assert: memory could not be allocated, %s\n", str);
		exit(1);
	}
}

#define MAX_NUM_THREADS ((unsigned long long)1 << 20)

void decode(const char *sourceFilename, const char *destinationFilename, unsigned long long codeLengthLimit, unsigned long long subBlockSize, unsigned long long originalFileSize, unsigned long long sizeOfEncodedData);

__global__ void decodeKernel(unsigned long long *subBlockInputBitOffsets, unsigned char *encodedData,
		unsigned char *decodedData, unsigned short *decodingTable,
		unsigned long long blockInputByteOffset, unsigned long long numSymbolsPerSubBlock,
		unsigned long long encodedDataBitSize, unsigned long long codeBitLengthLimit,
		unsigned long long blockSize)
{
//	__shared__ unsigned short sharedDecodingTable[1 << 12];
//	unsigned short decodingTableSize = 1 << codeBitLengthLimit;
//	for (unsigned i = threadIdx.x; i < decodingTableSize; i += blockDim.x) {
//		sharedDecodingTable[i] = decodingTable[i];
//	}
	
//	__syncthreads();
	
	unsigned long long subBlock = blockIdx.x*blockDim.x + threadIdx.x;
	if (subBlock >= blockSize)
		return;
	unsigned long long subBlockInputBitOffset = subBlockInputBitOffsets[subBlock] - 8*blockInputByteOffset;
	unsigned long long inputBitCurOffset = subBlockInputBitOffset;
	
	for (size_t codeWordIndex = 0;
		codeWordIndex < numSymbolsPerSubBlock
			& inputBitCurOffset < encodedDataBitSize - 8*blockInputByteOffset;
		codeWordIndex++)
	{
		unsigned long long byte = inputBitCurOffset/8;
		unsigned long long bit = inputBitCurOffset - 8*byte;
		unsigned long long code = 0 << 8*sizeof(unsigned long long);
		code |= (unsigned long long)encodedData[byte];
		code |= (unsigned long long)encodedData[byte+1] << 8;
		code |= (unsigned long long)encodedData[byte+2] << 16;
		code >>= bit;
		code &= 0xFFFFFFFFFFFFFFFFull >> (64 - codeBitLengthLimit);
		unsigned short decoderEntry = decodingTable[code];
		unsigned char output = (unsigned char)decoderEntry;
		unsigned long long length = decoderEntry >> 8;
		inputBitCurOffset += length;
		decodedData[subBlock * numSymbolsPerSubBlock + codeWordIndex] = output;
	}
}

bool debugMode = false;

int main(int argc, char** argv) {
	if (argc != 3 && argc != 4) {
		printf("Usage: ./decode <encodedFile> <destinationFile> <debug(optional)>\n");
		exit(0);
	}
	
	FILE *source = fopen(argv[1], "r");
	
	unsigned long long codeLengthLimit, subBlockSize, originalFileSize, sizeOfEncodedData;
	fread((void *)&codeLengthLimit, sizeof(unsigned long long), 1, source);
	printf("got here\n");
	fread((void *)&subBlockSize, sizeof(unsigned long long), 1, source);
	fread((void *)&originalFileSize, sizeof(unsigned long long), 1, source);
	fread((void *)&sizeOfEncodedData, sizeof(unsigned long long), 1, source);
	
	if (argc == 4) {
		debugMode = true;
		printf("Code Length Limit: %llu\n"
			"Sub-Block Size: %llu\n"
			"Original File Size: %llu\n"
			"Size of Encoded Data: %llu\n",
			codeLengthLimit,
			subBlockSize,
			originalFileSize,
			sizeOfEncodedData);
	}
	
	fclose(source);
	
	decode(argv[1], argv[2], codeLengthLimit, subBlockSize, originalFileSize, sizeOfEncodedData);
	
	return 0;
	
}

void printBits(unsigned long long n) {
	for (size_t i = 0; i < 8*sizeof(unsigned long long); ++i) {
		printf("%c", (n & ((unsigned long long)1 << i)) != (0 << 8*sizeof(unsigned long long)) ? '1' : '0');
	}
}

void printBits(unsigned char n) {
	for (size_t i = 0; i < 8*sizeof(unsigned char); ++i) {
		printf("%c", (n & ((unsigned char)1 << i)) != (0 << 8*sizeof(unsigned char)) ? '1' : '0');
	}
}

void printBits(unsigned short n) {
	for (size_t i = 0; i < 8*sizeof(unsigned short); ++i) {
		printf("%c", (n & ((unsigned short)1 << i)) != (0 << 8*sizeof(unsigned short)) ? '1' : '0');
	}
}

void decode(const char *sourceFilename, const char *destinationFilename, unsigned long long codeLengthLimit, unsigned long long subBlockSize, unsigned long long originalFileSize, unsigned long long sizeOfEncodedData) {
	
	hipError_t result;//for debugging
	
	unsigned long long subBlockOffsetTableSize = originalFileSize/subBlockSize + (originalFileSize%subBlockSize > 0);
	if (debugMode)
		printf("Offset Table Size: %llu\n", subBlockOffsetTableSize);
	unsigned long long decodeTableSize = (unsigned long long)1 << (codeLengthLimit + 1);
	if (debugMode)
		printf("Decoding Table Size: %llu\n", decodeTableSize);
	
	if (debugMode) printf("Reading decoding table...\n");
	FILE *decodingTableReader = fopen(sourceFilename, "r");
	fseek(decodingTableReader, subBlockOffsetTableSize*sizeof(unsigned long long) + 4*sizeof(unsigned long long), SEEK_SET);
	unsigned short *decodingTable = new unsigned short[decodeTableSize/2];
	assertSuccessfulAllocation(decodingTable, "decoding table");
	size_t numRead = fread((void *)decodingTable, sizeof(unsigned short), decodeTableSize/2, decodingTableReader);
	fclose(decodingTableReader);
	if (debugMode) printf("Finished reading decoding table.\n");
	
	unsigned long long offsetTableLocation = 8*4;
	if (debugMode) printf("Offset Table Location: %llu\n", offsetTableLocation);
	unsigned long long encodedDataLocation = 8*4 + 8*subBlockOffsetTableSize + decodeTableSize;
	if (debugMode) printf("Encoded Data Location: %llu\n", encodedDataLocation);
	
	FILE *offsetReader = fopen(sourceFilename, "r");
	fseek(offsetReader, offsetTableLocation, SEEK_SET);
	
	FILE *encodedDataReader = fopen(sourceFilename, "r");
	fseek(encodedDataReader, encodedDataLocation, SEEK_SET);
	if (debugMode) {
	//	unsigned char byte;
	//	while (fread((void *)&byte, sizeof(unsigned char), 1, encodedDataReader))
	//		printBits(byte);
	//	printf("\n");
	//	fseek(encodedDataReader, encodedDataLocation, SEEK_SET);
	}
	
	unsigned long long codeLengthLimitInBytes = codeLengthLimit/8 + (codeLengthLimit % 8 > 0);
	
	unsigned char *inputBuffer = new unsigned char[codeLengthLimitInBytes * MAX_NUM_THREADS * subBlockSize];
	assertSuccessfulAllocation(decodingTable, "decoding table");
	unsigned char *outputBuffer = new unsigned char[MAX_NUM_THREADS * subBlockSize];
	assertSuccessfulAllocation(outputBuffer, "output buffer");
	unsigned long long *offsetsBuffer = new unsigned long long[MAX_NUM_THREADS];
	assertSuccessfulAllocation(offsetsBuffer, "offsets buffer");
	
	if (debugMode) printf("Allocating CUDA memory...\n");
	unsigned char *cudaInput;
	result = hipMalloc(&cudaInput, codeLengthLimitInBytes * MAX_NUM_THREADS * subBlockSize);
	gpuErrorCheck(result);
	unsigned char *cudaOutput;
	result = hipMalloc(&cudaOutput, MAX_NUM_THREADS * subBlockSize);
	gpuErrorCheck(result);
	
	unsigned long long *cudaOffsets;
	result = hipMalloc(&cudaOffsets, MAX_NUM_THREADS * sizeof(unsigned long long));
	gpuErrorCheck(result);
	unsigned short *cudaDecodingTable;
	result = hipMalloc(&cudaDecodingTable, decodeTableSize);
	gpuErrorCheck(result);
	result = hipMemcpy(cudaDecodingTable, decodingTable, decodeTableSize, hipMemcpyHostToDevice);
	gpuErrorCheck(result);
	if (debugMode) printf("Finished allocating CUDA memory.\n");
	
	FILE *outputWriter = fopen(destinationFilename, "w");
	
	unsigned long long numSubBlocksRead = 0;
	unsigned long long numInputBytesRead = 0;
	while (numSubBlocksRead < subBlockOffsetTableSize) {
		unsigned long long numThreads = min((unsigned long long)MAX_NUM_THREADS, subBlockOffsetTableSize - numSubBlocksRead);
		if (debugMode) printf("Num Threads: %llu\n", numThreads);
		fread((void *)offsetsBuffer, sizeof(unsigned long long), numThreads, offsetReader);
		
		numInputBytesRead = offsetsBuffer[0]/8;
		fseek(encodedDataReader, encodedDataLocation + numInputBytesRead, SEEK_SET);
		unsigned long long numInputBytes = min(MAX_NUM_THREADS * subBlockSize * codeLengthLimitInBytes, (sizeOfEncodedData/8 + (sizeOfEncodedData%8 > 0)) - numInputBytesRead);
		if (debugMode) printf("Num Input Bytes Intended to be Read: %llu\n", numInputBytes);
		numRead = fread((void *)inputBuffer, sizeof(unsigned char), numInputBytes + 2, encodedDataReader);
		if (debugMode) {
			printf("Num Input Bytes Actually Read: %llu\n", numRead);
//			for (size_t i = 0; i < numInputBytes + 2; ++i)
//				printBits(inputBuffer[i]);
//			printf("\n");
		}
		
		result = hipMemcpy(cudaInput, inputBuffer, numInputBytes, hipMemcpyHostToDevice);
		gpuErrorCheck(result);
		result = hipMemcpy(cudaOffsets, offsetsBuffer, numThreads*sizeof(unsigned long long),
				hipMemcpyHostToDevice);
		gpuErrorCheck(result);
		
		dim3 threadsPerBlock(min((unsigned long long)512, subBlockOffsetTableSize));
		dim3 numBlocks(numThreads/threadsPerBlock.x + (numThreads%threadsPerBlock.x>0));
		if (debugMode) printf("Num Thread-Blocks in current kernel: %d\n", numBlocks.x);
		decodeKernel<<<numBlocks,threadsPerBlock>>>(
			cudaOffsets, cudaInput, cudaOutput, cudaDecodingTable,
			numInputBytesRead, subBlockSize, sizeOfEncodedData, codeLengthLimit, numThreads
		);
		
		result = hipMemcpy(outputBuffer, cudaOutput, numThreads*subBlockSize,
				hipMemcpyDeviceToHost);
		gpuErrorCheck(result);
		
		if (debugMode) {
			printf("Output bits for last 4 characters: ");
			printBits(outputBuffer[476*subBlockSize]);
			printBits(outputBuffer[476*subBlockSize + 1]);
			printBits(outputBuffer[476*subBlockSize + 2]);
			printBits(outputBuffer[476*subBlockSize + 3]);
			printf("\n");
			
			printf("Input offset for last block: ");
			printBits(offsetsBuffer[476]);
			printf("\n");
			
			printf("Input offset in decimal for last block: %llu\n", offsetsBuffer[476]);
			
			printf("Encoded data in input buffer: ");
			size_t offset = offsetsBuffer[476];
			printBits(inputBuffer[offset/8]);
			printBits(inputBuffer[offset/8 + 1]);
			printf("\n");
			printf("(First %llu bits above should be ignored.)\n", offset - 8*(offset/8));
			
			//printf("Decoding table result of encoded data:\n");
			unsigned long long index = inputBuffer[offset/8 + 1];
			index <<= 8;
			index |= inputBuffer[offset/8];
			index &= 0x0000000000000FFFull;
			printBits(index);
			printf("\n");
			printf("%c %d", (unsigned char)(decodingTable[index] & (unsigned short)0xFF), (int)(decodingTable[index] >> 8));
			printf("\n");
		}
		
		fwrite((void *)outputBuffer, sizeof(unsigned char),
			min(numThreads*subBlockSize, originalFileSize - numSubBlocksRead*subBlockSize),
			outputWriter);
		
		numSubBlocksRead += numThreads;
		if (debugMode) {
			static int x = 0;
			printf("Finished iteration %d\n", ++x);
		}
	}
	
	hipFree(cudaDecodingTable);
	hipFree(cudaInput);
	hipFree(cudaOutput);
	hipFree(cudaOffsets);
	
	delete [] offsetsBuffer;
	delete [] outputBuffer;
	delete [] inputBuffer;
	
}


